
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void cuda_add(float* a, float* b, float* c, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) { *c = *a + *b; }
}

extern "C" void add_arrays(float* a, float* b, float* c, int n) {
    float* da, *db, *dc;

    hipMalloc(&da, n*sizeof(float));
    hipMalloc(&db, n*sizeof(float));
    hipMalloc(&dc, n*sizeof(float));

    hipMemcpy(da, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n*sizeof(float), hipMemcpyHostToDevice);

    int block_size = 1024;
    int grid_size = (int)ceil((float)n/block_size);

    cuda_add<<<grid_size, block_size>>>(da, db, dc, n);

    hipMemcpy(c, dc, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
}
