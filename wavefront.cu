#include "hip/hip_runtime.h"
#define PRIME0 100030001
#define PRIME1 396191693

struct StateCounter {
    u32 extend_jobs;
    u32 shade_jobs;
};

struct PathStates {
    Ray* ray;
};

__global__ void generate_primary_rays(StateCounter* counter, PathStates* states, 
                                      PointCamera camera, u32 width, u32 height) 
{
    const u32 x = blockIdx.x * blockDim.x + threadIdx.x;
    const u32 y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const u32 id = y * width + x;
    const u32 seed = (id + framenum * PRIME0) * PRIME1;
    RngXor32 rng(seed);

    // nx and ny are in range (-1.0f, 1.0f)
    const f32 nx = (2.0f * cast(f32, x) + rng.random_f32()) / width  - 1.0f;
    const f32 ny = (2.0f * cast(f32, y) + rng.random_f32()) / height - 1.0f;

    const Ray primary_ray = camera.create_ray(nx, ny);

    states->ray[id] = primary_ray;

    atomicInc(&counter->extend_jobs);    
}

__global__ void extend_rays(StateCounter* counter, PathStates* states, 
                            BVHNode const * bvh, RenderTriangle const * triangles, 
                            u32 width, u32 height) 
{
    const u32 x = blockIdx.x * blockDim.x + threadIdx.x;
    const u32 y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const u32 id = y * width + x;
    const BVHTriangleIntersection isect = bvh_intersect_triangles(bvh, triangles, rays[id]);

    state.terminated = !isect.hit();

    atomicInc(counters.shade_jobs);
}

__global__ void shade(StateCounter* counter, PathStates* states) {
    
}



