#include "hip/hip_runtime.h"
#define PRIME0 100030001
#define PRIME1 396191693

struct PathStates {
    Ray* ray;
};

__global__ void generate_primary_rays(Ray* rays, PointCamera camera, u32 width, u32 height) {
    const u32 x = blockIdx.x * blockDim.x + threadIdx.x;
    const u32 y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const u32 id = y * width + x;
    const u32 seed = (id + framenum * PRIME0) * PRIME1;
    RngXor32 rng(seed);

    // nx and ny are in range (-1.0f, 1.0f)
    const f32 nx = (2.0f * cast(f32, x) + rng.random_f32()) / width  - 1.0f;
    const f32 ny = (2.0f * cast(f32, y) + rng.random_f32()) / height - 1.0f;

    const Ray primary_ray = camera.create_ray(nx, ny);

    rays[id] = primary_ray;
}

struct PathState {
    Ray      ray;
    RngXor32 rng;
};

__global__ void extend_rays(Ray* rays, BVHNode const * bvh, RenderTriangle const * triangles, u32 width, u32 height) {
    const u32 x = blockIdx.x * blockDim.x + threadIdx.x;
    const u32 y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const u32 id = y * width + x;
    const BVHTriangleIntersection isect = bvh_intersect_triangles(bvh, triangles, rays[id]);
}

__global__ void shade() {

}



