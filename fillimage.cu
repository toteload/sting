#include "hip/hip_runtime.h"
#include <stdint.h>
#include "vecmath.h"

surface<void, cudaSurfaceType2D> screen_surface;

__global__ void fill_screen_buffer(vec4* buffer, uint32_t width, uint32_t height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const int id = y * width + x;

    vec4 c = { 1.0f, 0.0f };
    c.r = float(x) / width;
    c.g = sinf(float(y)/height * 10);
    c.b = 0.0f;
    c.a = 1.0f;

    buffer[id] = c;
}

__global__ void blit_to_screen(vec4* buffer, uint32_t width, uint32_t height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const int id = y * width + x;

    surf2Dwrite<vec4>(buffer[id], screen_surface, x * sizeof(vec4), y, hipBoundaryModeZero);
}

void draw_test_image(hipArray_const_t array, vec4* screen_buffer, uint32_t width, uint32_t height) {
    cudaBindSurfaceToArray(screen_surface, array);
    dim3 threads = dim3(16, 16, 1);
    dim3 blocks = dim3((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y, 1);
    fill_screen_buffer<<<blocks, threads>>>(screen_buffer, width, height);
    blit_to_screen<<<blocks, threads>>>(screen_buffer, width, height);
    hipDeviceSynchronize();
}
