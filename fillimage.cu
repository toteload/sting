#include "hip/hip_runtime.h"
#include <stdint.h>
#include "vecmath.h"
#include "bvh.h"
#include "bvh.cpp"

surface<void, cudaSurfaceType2D> screen_surface;

__device__ bool intersect(BVHNode const * bvh, RenderTriangle const * triangles, Ray ray, HitRecord* hit_out) {
    float t;
    uint32_t tri_id;
    uint32_t aabb_isect_count;
    uint32_t tri_isect_count;
    const bool hit = bvh_intersect_triangles(bvh, triangles, ray, &t, &tri_id, &aabb_isect_count, &tri_isect_count);

    if (!hit) {
        return false;
    }

    const RenderTriangle& tri = triangles[tri_id];
    vec3 normal = triangle_normal(tri.v0, tri.v1, tri.v2);
    if (dot(normal, ray.dir) > 0.0f) {
        normal = -1.0f * normal;
    }

    HitRecord rec;
    rec.pos = ray.pos + t * ray.dir;
    rec.t = t;
    rec.normal = normal;

    *hit_out = rec;

    return true;
}

__global__ void fill_screen_buffer(BVHNode const * bvh, RenderTriangle const * triangles, PointCamera camera, 
                                   vec4* buffer, uint32_t width, uint32_t height) 
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const int id = y * width + x;

    // nx and ny are in range (-1.0f, 1.0f)
    const float nx = (2.0f * float(x) + 0.5f) / width  - 1.0f;
    const float ny = (2.0f * float(y) + 0.5f) / height - 1.0f;

    Ray ray = camera.create_ray(nx, ny);

    const vec3 point_light = { 0.0f, 1000.0f, 0.0f };

    vec4 c = vec4(0.0f, 0.0f, 0.0f, 1.0f);

#if 0
    /*
    // 4968
    for (uint32_t i = 0; i < 1000; i++) {
        float t;
        if (triangle_intersect(ray, triangles[i].v0, triangles[i].v1, triangles[i].v2, &t)) {
            c = vec4(1.0f, 0.0, 0.0f, 1.0f);
            break;
        }
    }
    */

    //float t;
    //uint32_t tri_id;
    //uint32_t aabb_isect_count;
    //uint32_t tri_isect_count;
    //const bool hit = bvh_intersect_triangles(bvh, triangles, ray, &t, &tri_id, &aabb_isect_count, &tri_isect_count);
    //float v = float(aabb_isect_count) / 80.0f;
    //float v = float(tri_isect_count) / 400.0f;
    //float v = hit ? 1.0f : 0.0f;
    c = vec4(v, v, v, 1.0f);
    //if (hit) { c = vec4(1.0f, 0.0f, 0.0f, 1.0f); }

    /*
    HitRecord rec;
    const bool hit = intersect(bvh, triangles, ray, &rec);
    if (hit) { c = vec4(1.0f, 0.0f, 0.0f, 1.0f); }
    */
#else
    HitRecord rec;
    const bool hit = intersect(bvh, triangles, ray, &rec);

    if (hit) {
        const float SHADOW_OFFSET_EPSILON = 0.0001f;

        const vec3 to_light = (point_light - rec.pos).normalize();
        const Ray shadow_ray = { rec.pos + SHADOW_OFFSET_EPSILON * rec.normal, to_light };

        const float max_distance = (point_light - rec.pos).length();

        const bool occluded = bvh_intersect_triangles_shadowcast(bvh, triangles, shadow_ray, max_distance);
        if (!occluded) {
            const float v = dot(to_light, rec.normal);
            c = vec4(v, v, v, 1.0f);
        }
    }
#endif

    buffer[id] = c;
}

__global__ void blit_to_screen(vec4* buffer, uint32_t width, uint32_t height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const int id = y * width + x;

    surf2Dwrite<vec4>(buffer[id], screen_surface, x * sizeof(vec4), y, hipBoundaryModeZero);
}

// ----------------------------------------------------------------------------

void render(BVHNode const * bvh, RenderTriangle const * triangles, PointCamera camera, 
            vec4* screen_buffer, uint32_t width, uint32_t height) 
{
    dim3 threads = dim3(16, 16, 1);
    dim3 blocks = dim3((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y, 1);
    fill_screen_buffer<<<blocks, threads>>>(bvh, triangles, camera, screen_buffer, width, height);
}

void render_buffer_to_screen(hipArray_const_t array, vec4* screen_buffer, uint32_t width, uint32_t height) {
    const dim3 threads = dim3(16, 16, 1);
    const dim3 blocks = dim3((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y, 1);

    cudaBindSurfaceToArray(screen_surface, array);
    blit_to_screen<<<blocks, threads>>>(screen_buffer, width, height);

    // Need to synchronize here otherwise it is very choppy
    hipDeviceSynchronize();
}
